#include "hip/hip_runtime.h"
/*
* Copyright (c) 2019, NVIDIA CORPORATION.  All rights reserved.
*
* NVIDIA CORPORATION and its licensors retain all intellectual property
* and proprietary rights in and to this software, related documentation
* and any modifications thereto.  Any use, reproduction, disclosure or
* distribution of this software and related documentation without an express
* license agreement from NVIDIA CORPORATION is strictly prohibited.
*/

#include <chrono>
#include <getopt.h>
#include <iostream>
#include <string>
#include <deque>
#include <mutex>
#include <future>
#include <thread>
#include <atomic>
#include <map>

#include "ThreadPool.h"

#include <claragenomics/logging/logging.hpp>
#include <claragenomics/io/fasta_parser.hpp>
#include <claragenomics/utils/cudautils.hpp>

#include <claragenomics/cudamapper/index.hpp>
#include <claragenomics/cudamapper/matcher.hpp>
#include <claragenomics/cudamapper/overlapper.hpp>
#include "overlapper_triggered.hpp"

static struct option options[] = {
    {"window-size", required_argument, 0, 'w'},
    {"kmer-size", required_argument, 0, 'k'},
    {"num-devices", required_argument, 0, 'd'},
    {"index-size", required_argument, 0, 'i'},
    {"target-index-size", required_argument, 0, 't'},
    {"max-cache-size", required_argument, 0, 'c'},
    {"help", no_argument, 0, 'h'},
};

void help(int32_t exit_code);

int main(int argc, char* argv[])
{
    claragenomics::logging::Init();

    uint32_t k               = 15;
    uint32_t w               = 15;
    size_t index_size        = 10000;
    size_t num_devices       = 1;
    size_t target_index_size = 10000;
    size_t max_cache_size    = 100;
    std::string optstring    = "t:i:k:w:h:d:c:";
    uint32_t argument;
    while ((argument = getopt_long(argc, argv, optstring.c_str(), options, nullptr)) != -1)
    {
        switch (argument)
        {
        case 'k':
            k = atoi(optarg);
            break;
        case 'w':
            w = atoi(optarg);
            break;
        case 'i':
            index_size = atoi(optarg);
            break;
        case 'd':
            num_devices = atoi(optarg);
            break;
        case 't':
            target_index_size = atoi(optarg);
            break;
        case 'c':
            max_cache_size = atoi(optarg);
            break;
        case 'h':
            help(0);
        default:
            exit(1);
        }
    }

    if (k > claragenomics::cudamapper::Index::maximum_kmer_size())
    {
        std::cerr << "kmer of size " << k << " is not allowed, maximum k = " << claragenomics::cudamapper::Index::maximum_kmer_size() << std::endl;
        exit(1);
    }

    // Check remaining argument count.
    if ((argc - optind) < 2)
    {
        std::cerr << "Invalid inputs. Please refer to the help function." << std::endl;
        help(1);
    }

    std::string query_filepath  = std::string(argv[optind++]);
    std::string target_filepath = std::string(argv[optind++]);

    bool all_to_all = false;
    if (query_filepath == target_filepath)
    {
        all_to_all        = true;
        target_index_size = index_size;
        std::cerr << "NOTE - Since query and target files are same, activating all_to_all mode. Query index size used for both files." << std::endl;
    }

    std::unique_ptr<claragenomics::io::FastaParser> query_parser = claragenomics::io::create_fasta_parser(query_filepath);
    int32_t queries                                              = query_parser->get_num_seqences();

    std::unique_ptr<claragenomics::io::FastaParser> target_parser = claragenomics::io::create_fasta_parser(target_filepath);
    int32_t targets                                               = target_parser->get_num_seqences();

    std::cerr << "Query " << query_filepath << " index " << queries << std::endl;
    std::cerr << "Target " << target_filepath << " index " << targets << std::endl;

    // Data structure for holding overlaps to be written out
    std::mutex overlaps_writer_mtx;

    struct query_target_range
    {
        std::pair<std::int32_t, int32_t> query_range;
        std::vector<std::pair<std::int32_t, int32_t>> target_ranges;
    };

    //First generate all the ranges independently, then loop over them.
    std::vector<query_target_range> query_target_ranges;

    for (std::int32_t query_start_index = 0; query_start_index < queries; query_start_index += index_size)
    {

        std::int32_t query_end_index = std::min(query_start_index + index_size, static_cast<size_t>(queries));

        query_target_range q;
        q.query_range = std::make_pair(query_start_index, query_end_index);

        std::int32_t target_start_index = 0;
        // If all_to_all mode, then we can optimzie by starting the target sequences from the same index as
        // query because all indices before the current query index are guaranteed to have been processed in
        // a2a mapping.
        if (all_to_all)
        {
            target_start_index = query_start_index;
        }

        for (; target_start_index < targets; target_start_index += target_index_size)
        {
            std::int32_t target_end_index = std::min(target_start_index + target_index_size,
                                                     static_cast<size_t>(targets));
            q.target_ranges.push_back(std::make_pair(target_start_index, target_end_index));
        }

        query_target_ranges.push_back(q);
    }

    // This is a per-device cache, if it has the index it will return it, if not it will generate it, store and return it.
    std::vector<std::map<std::pair<uint64_t, uint64_t>, std::shared_ptr<claragenomics::cudamapper::Index>>> index_cache(num_devices);

    auto get_index = [&index_cache, &max_cache_size](claragenomics::io::FastaParser& parser,
                                                     const claragenomics::cudamapper::read_id_t query_start_index,
                                                     const claragenomics::cudamapper::read_id_t query_end_index,
                                                     const std::uint64_t k,
                                                     const std::uint64_t w,
                                                     int device_id) {

        std::pair<uint64_t, uint64_t> key;
        key.first  = query_start_index;
        key.second = query_end_index;

        std::shared_ptr<claragenomics::cudamapper::Index> index;

        if (index_cache[device_id].count(key))
        {
            index = index_cache[device_id][key];
        }
        else
        {
	    auto index_unique_ptr = claragenomics::cudamapper::Index::create_index(parser, query_start_index, query_end_index,k,w);
            std::shared_ptr<claragenomics::cudamapper::Index> index(std::move(index_unique_ptr));

	    if (index_cache[device_id].size() < max_cache_size)
            {
                index_cache[device_id][key] = index;
            }
        }
        return index;
    };

    auto evict_index = [&index_cache](
                           const claragenomics::cudamapper::read_id_t query_start_index,
                           const claragenomics::cudamapper::read_id_t query_end_index,
                           int device_id) {

        std::pair<uint64_t, uint64_t> key;
        key.first  = query_start_index;
        key.second = query_end_index;

        index_cache[device_id].erase(key);
    };

    auto compute_overlaps = [&](query_target_range query_target_range, int device_id) {

        std::vector<std::shared_ptr<std::future<void>>> print_pafs_futures;

        hipSetDevice(device_id);

        auto query_start_index = query_target_range.query_range.first;
        auto query_end_index   = query_target_range.query_range.second;

        std::cerr << "Procecssing query range: (" << query_start_index << " - " << query_end_index - 1 << ")" << std::endl;

        std::shared_ptr<claragenomics::cudamapper::Index> query_index(nullptr);
        std::shared_ptr<claragenomics::cudamapper::Index> target_index(nullptr);
        std::unique_ptr<claragenomics::cudamapper::Matcher> matcher(nullptr);

        {
            CGA_NVTX_RANGE(profiler, "generate_query_index");
            auto start_time = std::chrono::high_resolution_clock::now();

            query_index = get_index(*query_parser, query_start_index, query_end_index, k, w, device_id);
        }

        //Main loop
        for (auto target_range : query_target_range.target_ranges)
        {

            auto target_start_index = target_range.first;
            auto target_end_index   = target_range.second;

            {
                CGA_NVTX_RANGE(profiler, "generate_target_index");
                auto start_time = std::chrono::high_resolution_clock::now();
                target_index    = get_index(*target_parser, target_start_index, target_end_index, k, w, device_id);
            }
            {
                CGA_NVTX_RANGE(profiler, "generate_matcher");
                auto start_time = std::chrono::high_resolution_clock::now();
                matcher         = claragenomics::cudamapper::Matcher::create_matcher(*query_index,
                                                                                     *target_index);
            }
            {

                claragenomics::cudamapper::OverlapperTriggered overlapper;
                CGA_NVTX_RANGE(profiler, "generate_overlaps");
                auto start_time = std::chrono::high_resolution_clock::now();

                // Get unfiltered overlaps
                std::vector<claragenomics::cudamapper::Overlap> overlaps_to_add;
                overlapper.get_overlaps(overlaps_to_add, matcher->anchors(), *query_index, *target_index);

                std::shared_ptr<std::future<void>> f = std::make_shared<std::future<void>>(std::async(std::launch::async, [&overlaps_writer_mtx, overlaps_to_add](std::vector<claragenomics::cudamapper::Overlap> overlaps) {
                    std::vector<claragenomics::cudamapper::Overlap> filtered_overlaps;
                    claragenomics::cudamapper::Overlapper::filter_overlaps(filtered_overlaps, overlaps_to_add);
                    overlaps_writer_mtx.lock();
                    claragenomics::cudamapper::Overlapper::print_paf(filtered_overlaps);
                    overlaps_writer_mtx.unlock();
                },
                                                                                                      overlaps_to_add));

                print_pafs_futures.push_back(f);
            }
        }
        //Query will no longer be needed on device, remove it from the cache
        evict_index(query_start_index, query_end_index, device_id);
        return print_pafs_futures;
    };

    // create thread pool to compute overlaps. One worker thread per device.
    ThreadPool overlap_pool(num_devices);

    // Enqueue the query-target ranges which need to be computed, each thread returns a vector of futures for the threads it launches
    std::vector<std::future<std::vector<std::shared_ptr<std::future<void>>>>> overlap_futures;
    for (int i = 0; i < query_target_ranges.size(); i++)
    {
        // enqueue and store future
        auto query_target_range = query_target_ranges[i];
        auto device_id          = i % num_devices;
        overlap_futures.push_back(overlap_pool.enqueue(compute_overlaps, query_target_range, device_id));
    }

    for (auto& f : overlap_futures)
    {
        for (auto a : f.get())
        {
            a->wait();
        }
    }

    return 0;
}

void help(int32_t exit_code = 0)
{
    std::cerr <<
        R"(Usage: cudamapper [options ...] <query_sequences> <target_sequences>
     <sequences>
        Input file in FASTA/FASTQ format (can be compressed with gzip)
        containing sequences used for all-to-all overlapping
     options:
        -k, --kmer-size
            length of kmer to use for minimizers [15] (Max=)"
              << claragenomics::cudamapper::Index::maximum_kmer_size() << ")"
              << R"(
        -w, --window-size
            length of window to use for minimizers [15])"
              << R"(
        -d, --num-devices
            number of GPUs to use [1])"
              << R"(
        -c, --max_cache_size
            number of indices to keep in GPU memory [100])"
              << R"(
        -i, --index-size
            length of batch size used for query [10000])"
              << R"(
        -t --target-index-size
            length of batch sized used for target [10000])"
              << std::endl;

    exit(exit_code);
}
